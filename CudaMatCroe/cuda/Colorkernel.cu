#include "hip/hip_runtime.h"
﻿#include "Kernel.hpp"
#include "Colorkernel.cuh"

__global__ void RGB2XYZ_Stream_Kernel(uint8_t* src, float* dst, uint32_t pixnumber, int MAX_BLOCK_UNIT)
{
	uint32_t id_x = threadIdx.x + MAX_BLOCK_UNIT * blockIdx.x;

	if (id_x > pixnumber)
	{
		return;
	}
	// pixel offset
	uint32_t p = MAX_BLOCK_UNIT * blockIdx.x * id_x;

	// RGBA offset
	uint8_t* prgba_cur = src + (p * 3);

	// LAB offset
	float* plab_cur = dst + (p * 3);

	// R
	float RR = (float)prgba_cur[0] / 255.0f;
	RR = RR > 0.04045 ? powf((RR + 0.055f) / 1.055f, 2.4f) : (RR / 12.92);
	// G
	float GG = (float)prgba_cur[1] / 255.0f;
	GG = GG > 0.04045 ? powf((GG + 0.055f) / 1.055f, 2.4f) : (GG / 12.92);
	// B
	float BB = (float)prgba_cur[2] / 255.0f;
	BB = BB > 0.04045 ? powf((BB + 0.055f) / 1.055f, 2.4f) : (BB / 12.92);
	// X
	plab_cur[0] = 0.4124564f * RR + 0.3575761f * GG + 0.1804375f * BB;
	// Y
	plab_cur[1] = 0.2126729f * RR + 0.7151522f * GG + 0.0721750f * BB;
	// Z
	plab_cur[2] = 0.0193339f * RR + 0.1191920f * GG + 0.9503041f * BB;
}

__global__ void XYZ2LAB_Stream_Kernel(float* src, float* dst, uint32_t pixnumber, int MAX_BLOCK_UNIT)
{
	uint32_t id_x = threadIdx.x + MAX_BLOCK_UNIT * blockIdx.x;

	if (id_x > pixnumber)
	{
		return;
	}

	// pixel offset
	uint32_t p = blockIdx.x * MAX_BLOCK_UNIT + id_x;

	// XYZ offset
	float* pxyz_cur = src + (p * 3);

	// LAB offset
	float* plab_cur = dst + (p * 3);

	// XYZ -> LAB
	const float param_13 = 1.0f / 3.0f;
	const float param_16116 = 16.0f / 116.0f;
	const float Xn = 0.950456f;
	const float Yn = 1.0f;
	const float Zn = 1.088754f;

	float fX, fY, fZ;

	float X = pxyz_cur[0] / (Xn);
	float Y = pxyz_cur[1] / (Yn);
	float Z = pxyz_cur[2] / (Zn);

	if (Y > 0.008856f)
		fY = pow(Y, param_13);
	else
		fY = 7.787f * Y + param_16116;

	if (X > 0.008856f)
		fX = pow(X, param_13);
	else
		fX = 7.787f * X + param_16116;

	if (Z > 0.008856)
		fZ = pow(Z, param_13);
	else
		fZ = 7.787f * Z + param_16116;

	float L = 116.0f * fY - 16.0f;
	L = L > 0.0f ? L : 0.0f;
	float A = 500.0f * (fX - fY);
	float B = 200.0f * (fY - fZ);

	plab_cur[0] = L;
	plab_cur[1] = A;
	plab_cur[2] = B;
}

__global__ void RGBA2RGB_Stream_Kernel(uint8_t* src, uint8_t* dst, uint32_t pixnumber, int MAX_BLOCK_UNIT)
{
	// pixel offset
	uint32_t id_x = threadIdx.x + MAX_BLOCK_UNIT * blockIdx.x;

	if (id_x > pixnumber)
	{
		return;
	}

	// RGBA offset
	uint8_t* prgba_cur = src + (id_x * 4);
	// RGB offset
	uint8_t* prgb_cur = dst + (id_x * 3);
	
	// copy pixel 
	prgb_cur[0] = prgba_cur[0];
	prgb_cur[1] = prgba_cur[1];
	prgb_cur[2] = prgba_cur[2];
}

__global__ void RGB2RBGA_Stream_Kernel(uint8_t* src, uint8_t* dst, uint32_t pixnumber, int MAX_BLOCK_UNIT)
{
	// pixel offset
	uint32_t id_x = threadIdx.x + MAX_BLOCK_UNIT * blockIdx.x;

	if (id_x > pixnumber)
	{
		return;
	}

	// RGBA offset
	uint8_t* prgba_cur = src + (id_x * 3);
	// RGB offset
	uint8_t* prgb_cur = dst + (id_x * 4);

	// copy pixel 
	prgba_cur[0] = prgb_cur[0];
	prgba_cur[1] = prgb_cur[1];
	prgba_cur[2] = prgb_cur[2];
	prgba_cur[3] = UCHAR_MAX;
}

bool gpuXYZ2LAB(float* src, float* dst, IMAGES_DATA_SEGMENT gpu_setting)
{
	XYZ2LAB_Stream_Kernel <<< gpu_setting.all_pixel_need_block, gpu_setting.MAX_BLOCK_HAS_THREAD_NUMBER >>> (src, dst, gpu_setting.all_image_pixel_number, gpu_setting.MAX_BLOCK_HAS_THREAD_NUMBER);
	return GetKernelFuntionStatus();
}

bool gpuRGB2XYZ(uint8_t* src, float* dst, IMAGES_DATA_SEGMENT gpu_setting)
{
	RGB2XYZ_Stream_Kernel <<< gpu_setting.all_pixel_need_block, gpu_setting.MAX_BLOCK_HAS_THREAD_NUMBER >>> (src, dst, gpu_setting.all_image_pixel_number, gpu_setting.MAX_BLOCK_HAS_THREAD_NUMBER);
	return GetKernelFuntionStatus();
}

bool gpuRGBA2RGB(uint8_t* src, uint8_t* dst, IMAGES_DATA_SEGMENT gpu_setting)
{
	RGBA2RGB_Stream_Kernel <<< gpu_setting.all_pixel_need_block, gpu_setting.MAX_BLOCK_HAS_THREAD_NUMBER >>> (src, dst, gpu_setting.all_image_pixel_number, gpu_setting.MAX_BLOCK_HAS_THREAD_NUMBER);
	return GetKernelFuntionStatus();
}